#include "hip/hip_runtime.h"
#include <stdio.h>


int main(void) {
  hipDeviceProp_t prop;

  int count;
  hipGetDeviceCount(&count);

  printf("Device count: %d\n", count);
  
  for(int i = 0; i < count; ++i) {
    hipGetDeviceProperties(&prop, i);

    printf("Device's name(%d): %s\n", i, prop.name);
    printf("  Total global mem: %zu\n", prop.totalGlobalMem);
    printf("  Shared Mem per Block: %zu\n", prop.sharedMemPerBlock);
    printf("  Register per block: %d\n", prop.regsPerBlock);
    printf("  Warp Size: %d\n", prop.warpSize);
    printf("  Mem Pitch: %zu\n", prop.memPitch);
    printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("  Max Threads dim[0]: %d\n", prop.maxThreadsDim[0]);
    printf("  Max Threads dim[1]: %d\n", prop.maxThreadsDim[1]);
    printf("  Max Threads dim[2]: %d\n", prop.maxThreadsDim[2]);
    printf("  Max Grid Size [0]: %d\n", prop.maxGridSize[0]);
    printf("  Max Grid Size [1]: %d\n", prop.maxGridSize[1]);
    printf("  Max Grid Size [2]: %d\n", prop.maxGridSize[2]);
    printf("  Total Const Mem: %zu\n", prop.totalConstMem);
    printf("  Major: %d\n", prop.major);
    printf("  Minor: %d\n", prop.minor);
    printf("  Texture Alignment: %zu\n", prop.textureAlignment);
    printf("  Device Overlap: %d\n", prop.deviceOverlap);
    printf("  MultiProcessor Count: %d\n", prop.multiProcessorCount);
    printf("  Kernel Exec Timeout Enabled: %d\n", prop.kernelExecTimeoutEnabled);
    printf("  Integrated: %d\n", prop.integrated);
    printf("  Can Map Host Memory: %d\n", prop.canMapHostMemory);
    printf("  Compute Mode: %d\n", prop.computeMode);
    printf("  Max Texture 1D: %d\n", prop.maxTexture1D);
    printf("  Max Texture 2D [0]: %d\n", prop.maxTexture2D[0]);
    printf("  Max Texture 2D [1]: %d\n", prop.maxTexture2D[1]);
    printf("  Max Texture 3D [0]: %d\n", prop.maxTexture3D[0]);
    printf("  Max Texture 3D [1]: %d\n", prop.maxTexture3D[1]);
    printf("  Max Texture 3D [2]: %d\n", prop.maxTexture3D[2]);
    printf("  Concurrent Kernels: %d\n", prop.concurrentKernels);
  }

  return 0;
}
